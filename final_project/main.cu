#include <stdio.h>
#include <iostream>
#include <fstream>
#include "kernel.cu"
#include "support.cu"
#include "launcher.cu"

int main (int argc, char *argv[]) {
    // read kernel config file
    FILE *f = fopen("config/config_1.json", "rb");
    fseek(f, 0, SEEK_END);
    long fsize = ftell(f);
    fseek(f, 0, SEEK_SET);
    char *fc; 
    fc = (char *)malloc((fsize+1) * sizeof(char));
    fread(fc, 1, fsize, f);
    fclose(f);
    fc[fsize] = 0;

    scheduler(fc, NULL, 0);

    // hipMemcpy(output, in_d, n * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("# Done!\n");
}