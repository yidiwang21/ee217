#include "hip/hip_runtime.h"
#ifndef __KERNEL_CU__
#define __KERNEL_CU__

#include <stdint.h>
#include "support.cu"

#define ITER_NUM    10000   // just an amplifier

// My kernels do something meaningless like reversing content in shared memory
// just spin for a while and allocate some shared memory

typedef struct {
    int *input;
    int input_num;
}InputStruct;

__global__ void kernel_basic_reverse(int *d, int n, int delay) {
    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();
}

__global__ void kernel_shared_mem_1024(int *d, int n, int delay) {
    __shared__ uint8_t SharedMemArr[1024];
    int t = threadIdx.x;
    int tr = n - t - 1;
    SharedMemArr[t] = d[t];
    __syncthreads();
    
    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();

    d[t] = SharedMemArr[tr];
}

__global__ void kernel_shared_mem_4096(int *d, int n, int delay) {
    __shared__ uint8_t SharedMemArr[4096];
    int t = threadIdx.x;
    int tr = n - t - 1;
    SharedMemArr[t] = d[t];
    __syncthreads();
    
    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();

    d[t] = SharedMemArr[tr];
}

__global__ void kernel_shared_mem_16384(int *d, int n, int delay) {
    __shared__ uint8_t SharedMemArr[16384];
    int t = threadIdx.x;
    int tr = n - t - 1;
    SharedMemArr[t] = d[t];
    __syncthreads();
    
    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();

    d[t] = SharedMemArr[tr];
}

// TODO: compute exe time for each block

__global__ void lazyKernel_0(int delay) {
    int bytes_per_thread = 0;
    for (int j = 0; j < ITER_NUM; j++) {
        bytes_per_thread = j;
    }

    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();
}

__global__ void lazyKernel_1024(int delay) {
    __shared__ uint8_t SharedMemArr[1024];
 
    int bytes_per_thread = 1024 / blockDim.x;
    for (int j = 0; j < ITER_NUM; j++) {
        for (unsigned int i = 0; i < bytes_per_thread; i++) 
            SharedMemArr[bytes_per_thread * threadIdx.x + i] = threadIdx.x;
    }
    
    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();
}

__global__ void lazyKernel_4096(int delay) {
    __shared__ uint8_t SharedMemArr[4096];
 
    int bytes_per_thread = 4096 / blockDim.x;
    for (int j = 0; j < ITER_NUM; j++) {
        for (unsigned int i = 0; i < bytes_per_thread; i++) 
            SharedMemArr[bytes_per_thread * threadIdx.x + i] = threadIdx.x;
    }

    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();
}

__global__ void lazyKernel_8192(int delay) {
    __shared__ uint8_t SharedMemArr[8192];
 
    int bytes_per_thread = 8192 / blockDim.x;
    for (int j = 0; j < ITER_NUM; j++) {
        for (unsigned int i = 0; i < bytes_per_thread; i++) 
            SharedMemArr[bytes_per_thread * threadIdx.x + i] = threadIdx.x;
    }

    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();
}

__global__ void lazyKernel_16384(int delay) {
    __shared__ uint8_t SharedMemArr[16384];
 
    // do something boring!
    int bytes_per_thread = 16384 / blockDim.x;
    for (int j = 0; j < ITER_NUM; j++) {
        for (unsigned int i = 0; i < bytes_per_thread; i++) 
            SharedMemArr[bytes_per_thread * threadIdx.x + i] = threadIdx.x;
    }

    unsigned int start = kernelTimer();
    while (kernelTimer() - start < delay);
    unsigned int stop = kernelTimer();
}

#endif