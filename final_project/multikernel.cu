#include "multikernel.cuh"

// init MultiKernel class with the input arguments
MultiKernel::MultiKernel(char *config_file) {
    parser = cJSON_Parse(config_file);    
    if (parser == NULL) {
        fprintf(stderr, "# Invalid kernel config file!\n");
        cleanUp();
    }
}

Node* MultiKernel::newNode() {
    Node *new_node = (Node*)malloc(sizeof(Node));
    new_node->kernel_id = 0;

    return new_node;
}

Node* MultiKernel::splitNode(Node **node, int w, int h, int kid) {
    (*node)->used = 1;
    (*node)->kernel_id = kid;

    (*node)->left = newNode();
    (*node)->left->used = 1;
    (*node)->left->kernel_id = (*node)->kernel_id;
    (*node)->left->parent = (*node);
    (*node)->left->left = NULL;
    (*node)->left->right = NULL;
    (*node)->left->start_point.x = (*node)->start_point.x;
    (*node)->left->start_point.y = (*node)->start_point.y + h;
    // (*node)->left->end_point->x = (*node)->end_point->x;
    // (*node)->left->end_point->y = (*node)->end_point->;
    (*node)->left->width = (*node)->width;
    (*node)->left->height = (*node)->height - h;

    (*node)->right = newNode();
    (*node)->left->used = 0;
    (*node)->right->kernel_id = 0;
    (*node)->parent = (*node);
    (*node)->right->left = NULL;
    (*node)->right->right = NULL;
    (*node)->right->start_point.x = (*node)->start_point.x + w;
    (*node)->right->start_point.y = (*node)->start_point.y;
    (*node)->right->width = (*node)->width - w;
    (*node)->right->height = (*node)->height;

    return (*node);
}

Node* MultiKernel::findBestFit(Node *root, int w, int h) {
    if (root->used == 1) {
        Node *rightchild = findBestFit(root->right, w, h);
        if (rightchild != NULL) return rightchild;
        Node *leftchild = findBestFit(root->left, w, h);
        return leftchild;
    }else if ((w <= root->width) && (h <= root->height))
        return root;
    else
        return NULL;
}

void MultiKernel::kernelInfoInit() {
    cJSON *entry = NULL;
    cJSON *iter = NULL;
    hipGetDeviceProperties(&devProp, 0);

    // printf("# Parsing kernel config file...\n");
    // printf("[\n");

    entry = cJSON_GetObjectItem(parser, "kernel_number");
    if (!entry || entry->type != cJSON_Number) {
        fprintf(stderr, "# Invalid kernel number!\n");
        cleanUp();
    }
    kernel_num = entry->valueint;
    entry = NULL;
    entry = cJSON_GetObjectItem(parser, "sched_policy");
    if (!entry || entry->type != cJSON_Number || entry->valueint > 2 || entry->valueint < 0) {
        fprintf(stderr, "Invalid policy! (must be 1 ,2, 3)\n");
        cleanUp();
    }
    sched_policy = entry->valueint;
    entry = NULL;

    entry = cJSON_GetObjectItem(parser, "kernel_entry");
    if (!entry) {
        fprintf(stderr, "Invalid kernel entry!\n");
        cleanUp();
    }

    kernel_list = (KernelInfo*)malloc(kernel_num * sizeof(KernelInfo));
    
    int idx = 0;
    cJSON_ArrayForEach(iter, entry) {
        cJSON *grid_size_entry = cJSON_GetObjectItem(iter, "grid_size");
        cJSON *block_size_entry = cJSON_GetObjectItem(iter, "block_size");
        cJSON *shared_mem_entry = cJSON_GetObjectItem(iter, "shared_mem");
        cJSON *duration_entry = cJSON_GetObjectItem(iter, "duration");
        cJSON *kernel_id_entry = cJSON_GetObjectItem(iter, "kernel_id");

        kernel_list[idx].grid_size = grid_size_entry->valueint;
        kernel_list[idx].block_size = block_size_entry->valueint;   // must be in a range
        kernel_list[idx].duration = duration_entry->valueint;
        kernel_list[idx].shared_mem = shared_mem_entry->valueint;
        kernel_list[idx].kernel_id = kernel_id_entry->valueint;
        // printf("        grid size = %d\n", kernel_list[idx].grid_size);
        // printf("        block size = %d\n", kernel_list[idx].block_size);
        // printf("        duration = %d\n", kernel_list[idx].duration);
        // printf("        shared mem size = %d\n", kernel_list[idx].shared_mem);
        
        idx++;
    }
}

void MultiKernel::blockInfoInit() {
    count = 0;  // number of blocks averagely per SM
    for (int i = 0; i < kernel_num; i++) {
        for (int j = 0; j < kernel_list[i].grid_size/SM_NUM+1; j++) {
            count++;
        }
    }

    block_list = (BlockInfo*)malloc(sizeof(BlockInfo) * count);

    int idx = 0;
    for (int i = 0; i < kernel_num; i++) {
        for (int j = 0; j < (kernel_list[i].grid_size-1)/SM_NUM+1; j++) {
            block_list[idx+j].kernel_id = kernel_list[i].kernel_id;
            block_list[idx+j].block_size = kernel_list[i].block_size;
            block_list[idx+j].duration = kernel_list[i].duration;
        }
        idx += (kernel_list[i].grid_size-1)/SM_NUM+1;
    }
}

void MultiKernel::sortDurationDecending() {
    bool swapped;
    do {
        swapped = false;
        for (int i = 0; i < kernel_num - 1; i++) {
            if (kernel_list[i].duration < kernel_list[i+1].duration) {
                std::swap(kernel_list[i], kernel_list[i+1]);
                swapped = true; 
            }
        }
    } while(swapped);   
}

void MultiKernel::sortStartTimeAscending() {
    int tmp = 0;
    int c = 0;
    for (int i = 0; i < kernel_num; i++) {
        c = 0;
        tmp = 0;
        for (int j = 0; j < count; j++) {
            if (kernel_list[i].kernel_id == block_list[j].kernel_id) {
                if (c > 0) {
                    kernel_list[i].start_time = MIN(block_list[j].start_time, tmp);
                }else {
                    kernel_list[i].start_time = block_list[j].start_time;
                }
                tmp = kernel_list[i].start_time;
                c++;
            }
        }
    }

    bool swapped;
    do {
        swapped = false;
        for (int i = 0; i < kernel_num - 1; i++) {
            if (kernel_list[i].start_time > kernel_list[i+1].start_time) {
                std::swap(kernel_list[i], kernel_list[i+1]);
                swapped = true; 
            }
        }
    } while(swapped); 
}

void MultiKernel::scheduler() {
    sortDurationDecending();
    blockInfoInit();

    // Step 1: initialize resources
    Node *root = newNode();
    root->used = 0;
    root->height = devProp.maxThreadsPerMultiProcessor;
    root->width = INT_MAX;
    root->start_point.x = 0;
    root->start_point.y = 0;
    root->left = NULL;
    root->right = NULL;
    // Step 2: fit
    for (int i = 0; i < count; i++) {
        Node *node;
        Node *block_node;
        node = findBestFit(root, block_list[i].duration, block_list[i].block_size);
        if (node != NULL) {
            // assign a leaf to a block
            block_node = splitNode(&node, block_list[i].duration, block_list[i].block_size, block_list[i].kernel_id);
            block_list[i].start_time = block_node->start_point.x;
        }
    }
    sortStartTimeAscending();
}

// this should take sorted kernels as input
void MultiKernel::GPUResourceInit() {
    for (int i = 0; i < kernel_num; i++) {
        kernel_list[i].block_times = (uint64_t *)malloc(sizeof(uint64_t) * kernel_list[i].grid_size * 2);
        kernel_list[i].block_smids = (uint32_t *)malloc(sizeof(uint32_t) * kernel_list[i].grid_size);
        memset(kernel_list[i].block_times, 0, sizeof(kernel_list[i].block_times));
        memset(kernel_list[i].block_smids, 0, sizeof(kernel_list[i].block_smids));

        hipMalloc((void**) &kernel_list[i].block_times_d, sizeof(uint64_t) * kernel_list[i].grid_size * 2);
        hipMalloc((void**) &kernel_list[i].block_smids_d, sizeof(uint32_t) * kernel_list[i].grid_size);
    }
    hipDeviceSynchronize();
    for (int i = 0; i < kernel_num; i++) {
        hipMemcpy(kernel_list[i].block_times_d, kernel_list[i].block_times, sizeof(uint64_t) * kernel_list[i].grid_size * 2, hipMemcpyHostToDevice);
        hipMemcpy(kernel_list[i].block_smids_d, kernel_list[i].block_smids, sizeof(uint32_t) * kernel_list[i].grid_size, hipMemcpyHostToDevice);
    }
    hipDeviceSynchronize();
}

void MultiKernel::kernelLauncher() {
    // scheduler();
    kernelInfoInit();
    GPUResourceInit();

    hipError_t cuda_ret;

    hipStream_t *streams;
    streams = (hipStream_t*)malloc(kernel_num * sizeof(hipStream_t));
    for (int i = 0; i < kernel_num; i++) hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    hipDeviceSynchronize();

    printf("Launching kernel...\n"); fflush(stdout);
    printf("Kernel number: %d\n", kernel_num);
    for (int i = 0; i < kernel_num; i++) {
        GPUSpin <<<kernel_list[i].grid_size, kernel_list[i].block_size, 0, streams[i]>>> (kernel_list[i].duration, kernel_list[i].block_times_d, kernel_list[i].block_smids_d);
    }
    hipDeviceSynchronize();

    for (int i = 0; i < kernel_num; i++) {
        hipMemcpy(kernel_list[i].block_times, kernel_list[i].block_times_d, sizeof(uint64_t) * kernel_list[i].grid_size * 2, hipMemcpyDeviceToHost);
        hipMemcpy(kernel_list[i].block_smids, kernel_list[i].block_smids_d, sizeof(uint32_t) * kernel_list[i].grid_size, hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();

    // print
    for (int i = 0; i < kernel_num; i++) {
        printf("=========================================\n");
        for (int j = 0; j < kernel_list[i].grid_size; j++) {
            // nano sec timer to ms
            kernel_list[i].block_times[j*2] = (kernel_list[i].block_times[j*2] / 1000 / 1000) % 10000;
            kernel_list[i].block_times[j*2+1] = (kernel_list[i].block_times[j*2+1] / 1000 / 1000) % 10000;
            printf("Block index: %d\n", j);
            printf("kernel id: %d\n", i);
            printf("SM id: %d\n", kernel_list[i].block_smids[j]);
            printf("start time: %lu\n", kernel_list[i].block_times[j*2]);
            printf("stop time: %lu\n", kernel_list[i].block_times[j*2+1]);
            printf("elapsed time: %lu\n\n", kernel_list[i].block_times[j*2+1] - kernel_list[i].block_times[j*2]);
        }
    }
}

void MultiKernel::cleanUp() {
    cJSON_Delete(parser);
    free(kernel_list);
    exit(0);
}