
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float partialSum[2*BLOCK_SIZE];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    if (start + t < size) 
        partialSum[t] = in[start + t];
    else 
        partialSum[t] = 0;
    if (start + t + blockDim.x < size) 
        partialSum[blockDim.x + t] = in[start + blockDim.x + t];
    else 
        partialSum[blockDim.x + t] = 0;

    for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
        __syncthreads();
        // if (t < stride && t + start + stride < size) {
        if (t < stride) {
            partialSum[t] += partialSum[t + stride];
        }
    }

    if (t == 0)
        out[blockIdx.x] = partialSum[0];
}
