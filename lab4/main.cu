#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.cu"
#include "support.cu"

#define STREAM_NUM  3
#define BLOCK_SIZE  512

int main (int argc, char *argv[])
{
    srand(217);

    Timer timer;

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned VecSize;
    if (argc == 1) VecSize = 1000000;
    else if (argc == 2) VecSize = atoi(argv[1]);   
    else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
    }

    int SegSize = VecSize / STREAM_NUM;
    int offset = VecSize - STREAM_NUM * SegSize; // for the last partition
    const unsigned int GRID_SIZE = (int)ceil((float(SegSize + offset)) / BLOCK_SIZE);

    float *A, *B, *C;
    // A = (float*)malloc(VecSize * sizeof(float));
    // B = (float*)malloc(VecSize * sizeof(float));
    // C = (float*)malloc(VecSize * sizeof(float));
    hipHostAlloc((void **) &A, VecSize * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **) &B, VecSize * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **) &C, VecSize * sizeof(float), hipHostMallocDefault);
    for (int i = 0; i < VecSize; i++) A[i] = (rand()%100)/100.00;
    for (int i = 0; i < VecSize; i++) B[i] = (rand()%100)/100.00;
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    size Of vector: %u\n  ", VecSize);
    
    // partition vectors
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);
    float *d_A0, *d_B0, *d_C0;
    float *d_A1, *d_B1, *d_C1;
    float *d_A2, *d_B2, *d_C2;

    // float *d_A3, *d_B3, *d_C3;
    
    hipMalloc((void**) &d_A0, SegSize * sizeof(float));
    hipMalloc((void**) &d_B0, SegSize * sizeof(float));
    hipMalloc((void**) &d_C0, SegSize * sizeof(float));
    
    hipMalloc((void**) &d_A1, SegSize * sizeof(float));
    hipMalloc((void**) &d_B1, SegSize * sizeof(float));
    hipMalloc((void**) &d_C1, SegSize * sizeof(float));

    // used for 3 streams
    hipMalloc((void**) &d_A2, (offset + SegSize) * sizeof(float));
    hipMalloc((void**) &d_B2, (offset + SegSize) * sizeof(float));
    hipMalloc((void**) &d_C2, (offset + SegSize) * sizeof(float));

    // used for 4 streams
    // hipMalloc((void**) &d_A2, SegSize * sizeof(float));
    // hipMalloc((void**) &d_B2, SegSize * sizeof(float));
    // hipMalloc((void**) &d_C2, SegSize * sizeof(float));
    // hipMalloc((void**) &d_A3, (offset + SegSize) * sizeof(float));
    // hipMalloc((void**) &d_B3, (offset + SegSize) * sizeof(float));
    // hipMalloc((void**) &d_C3, (offset + SegSize) * sizeof(float));

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // create streams
    // FIXME:
    hipStream_t streams[STREAM_NUM];
    for (int i = 0; i < STREAM_NUM; i++) {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    hipMemcpyAsync(d_A0, A, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_B0, B, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_A1, A + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[1]);
    hipMemcpyAsync(d_B1, B + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[1]);
    // used for 3 streams
    hipMemcpyAsync(d_A2, A + 2 * SegSize, (offset + SegSize) * sizeof(float), hipMemcpyHostToDevice, streams[2]);
    hipMemcpyAsync(d_B2, B + 2 * SegSize, (offset + SegSize) * sizeof(float), hipMemcpyHostToDevice, streams[2]);
    // used for 4 streams
    // hipMemcpyAsync(d_A2, A + 2*SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[2]);
    // hipMemcpyAsync(d_B2, B + 2*SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[2]);
    // hipMemcpyAsync(d_A3, A + 3*SegSize, (offset + SegSize) * sizeof(float), hipMemcpyHostToDevice, streams[3]);
    // hipMemcpyAsync(d_B3, B + 3*SegSize, (offset + SegSize) * sizeof(float), hipMemcpyHostToDevice, streams[3]);
    
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[0]>>> (SegSize ,d_A0, d_B0, d_C0);
    VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[1]>>> (SegSize ,d_A1, d_B1, d_C1);
    // used for 3 streams
    VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[2]>>> (SegSize + offset ,d_A2, d_B2, d_C2);
    // used for 4 streams
    // VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[2]>>> (SegSize ,d_A2, d_B2, d_C2);
    // VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[3]>>> (SegSize + offset ,d_A3, d_B3, d_C3);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);
    hipMemcpyAsync(C, d_C0, SegSize * sizeof(float), hipMemcpyDeviceToHost, streams[0]);
    hipMemcpyAsync(C + SegSize, d_C1, SegSize * sizeof(float), hipMemcpyDeviceToHost, streams[1]);
    // used for 3 streams
    hipMemcpyAsync(C + 2 * SegSize, d_C2, (offset + SegSize) * sizeof(float), hipMemcpyDeviceToHost, streams[2]);
    // used for 4 streams
    // hipMemcpyAsync(C + 2*SegSize, d_C2, SegSize * sizeof(float), hipMemcpyDeviceToHost, streams[2]);
    // hipMemcpyAsync(C + 3 * SegSize, d_C3, (offset + SegSize) * sizeof(float), hipMemcpyDeviceToHost, streams[3]);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    printf("Verifying results..."); fflush(stdout);
    verify(A, B, C, VecSize);

    hipHostFree(A); 
    hipHostFree(B);
    hipHostFree(C);
    hipFree(d_A0);
    hipFree(d_B0);
    hipFree(d_C0);
    hipFree(d_A1);
    hipFree(d_B1);
    hipFree(d_C1);
    hipFree(d_A2);
    hipFree(d_B2);
    hipFree(d_C2);
    // hipFree(d_A3);
    // hipFree(d_B3);
    // hipFree(d_C3);

    return 0;

}