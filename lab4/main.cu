#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.cu"
#include "support.cu"

#define STREAM_NUM  3
#define BLOCK_SIZE  512

int main (int argc, char *argv[])
{
    srand(217);

    Timer timer;
    hipError_t cuda_ret;

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned VecSize;
    if (argc == 1) VecSize = 1000000;
    else if (argc == 2) VecSize = atoi(argv[1]);   
    else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
    }

    int SegSize = VecSize / STREAM_NUM;
    int offset = VecSize - STREAM_NUM * SegSize; // for the last partition
    const unsigned int GRID_SIZE = (int)ceil((float(SegSize + offset)) / BLOCK_SIZE);

    float *A, *B, *C;
    A = (float*)malloc(VecSize * sizeof(float));
    B = (float*)malloc(VecSize * sizeof(float));
    C = (float*)malloc(VecSize * sizeof(float));
    for (int i = 0; i < VecSize; i++) A[i] = (rand()%100)/100.00;
    for (int i = 0; i < VecSize; i++) B[i] = (rand()%100)/100.00;
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    size Of vector: %u x %u\n  ", VecSize);
    
    // partition vectors
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);
    float *d_A0, *d_B0, *d_C0;
    float *d_A1, *d_B1, *d_C1;
    float *d_A2, *d_B2, *d_C2;
    
    hipMalloc((void**) &d_A0, SegSize * sizeof(float));
    hipMalloc((void**) &d_B0, SegSize * sizeof(float));
    hipMalloc((void**) &d_C0, SegSize * sizeof(float));
    
    hipMalloc((void**) &d_A1, SegSize * sizeof(float));
    hipMalloc((void**) &d_B1, SegSize * sizeof(float));
    hipMalloc((void**) &d_C1, SegSize * sizeof(float));
    hipMalloc((void**) &d_A2, (offset + SegSize) * sizeof(float));
    hipMalloc((void**) &d_B2, (offset + SegSize) * sizeof(float));
    hipMalloc((void**) &d_C2, (offset + SegSize) * sizeof(float));

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // create streams
    // FIXME:
    hipStream_t streams[STREAM_NUM];
    for (int i = 0; i < STREAM_NUM; i++) {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    hipMemcpyAsync(d_A0, A, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_B0, B, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_A1, A + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[1]);
    hipMemcpyAsync(d_B1, B + SegSize, SegSize * sizeof(float), hipMemcpyHostToDevice, streams[1]);
    hipMemcpyAsync(d_A2, A + 2 * SegSize, (offset + SegSize) * sizeof(float), hipMemcpyHostToDevice, streams[2]);
    hipMemcpyAsync(d_B2, B + 2 * SegSize, (offset + SegSize) * sizeof(float), hipMemcpyHostToDevice, streams[2]);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[0]>>> (SegSize ,d_A0, d_B0, d_C0);
    VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[1]>>> (SegSize ,d_A1, d_B1, d_C1);
    VecAdd <<<GRID_SIZE, BLOCK_SIZE, 0, streams[2]>>> (SegSize + offset ,d_A2, d_B2, d_C2);
    cuda_ret = hipDeviceSynchronize();  
    if(cuda_ret != hipSuccess) { fprintf(stderr, "Unable to launch kernel!\n"); exit(-1); }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);
    hipMemcpyAsync(C, d_C0, SegSize * sizeof(float), hipMemcpyDeviceToHost, streams[0]);
    hipMemcpyAsync(C + SegSize, d_C1, SegSize * sizeof(float), hipMemcpyDeviceToHost, streams[1]);
    hipMemcpyAsync(C + 2 * SegSize, d_C2, (offset + SegSize) * sizeof(float), hipMemcpyDeviceToHost, streams[2]);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    printf("Verifying results..."); fflush(stdout);
    verify(A, B, C, VecSize);

    free(A); 
    free(B);
    free(C);
    hipFree(d_A0);
    hipFree(d_B0);
    hipFree(d_C0);
    hipFree(d_A1);
    hipFree(d_B1);
    hipFree(d_C1);
    hipFree(d_A2);
    hipFree(d_B2);
    hipFree(d_C2);

    return 0;

}
